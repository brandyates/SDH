/* ==================================================================
	Programmer: Yicheng Tu (ytu@cse.usf.edu)
	The basic SDH algorithm implementation for 3D data
	To compile: nvcc SDH.c -o SDH in the C4 lab machines

	CUDA Portion and kernels programmed by: Brandon Yates
   ==================================================================
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>


#define BOX_SIZE	23000 /* size of the data box on one dimension            */

/* descriptors for single atom in the tree */
typedef struct atomdesc {
	double x_pos;
	double y_pos;
	double z_pos;
} atom;

typedef struct hist_entry{
	//float min;
	//float max;
	unsigned long long d_cnt;   /* need a long long type as the count might be huge */
} bucket;


bucket * histogram;		/* list of all buckets in the histogram   */
long long	PDH_acnt;	/* total number of data points            */
int num_buckets;		/* total number of buckets in the histogram */
double   PDH_res;		/* value of w                             */
atom * atom_list;		/* list of all data points                */

/* These are for an old way of tracking time */
struct timezone Idunno;	
struct timeval startTime, endTime;


/* 
	distance of two points in the atom_list 
*/
double p2p_distance(int ind1, int ind2) {
	
	double x1 = atom_list[ind1].x_pos;
	double x2 = atom_list[ind2].x_pos;
	double y1 = atom_list[ind1].y_pos;
	double y2 = atom_list[ind2].y_pos;
	double z1 = atom_list[ind1].z_pos;
	double z2 = atom_list[ind2].z_pos;
		
	return sqrt((x1 - x2)*(x1-x2) + (y1 - y2)*(y1 - y2) + (z1 - z2)*(z1 - z2));
}

/*Kernel of p2p distance*/
__device__ double p2p_distance_kernel(atom* atom_list, int ind1, int ind2){
	double x1 = atom_list[ind1].x_pos;
	double x2 = atom_list[ind2].x_pos;
	double y1 = atom_list[ind1].y_pos;
	double y2 = atom_list[ind2].y_pos;
	double z1 = atom_list[ind1].z_pos;
	double z2 = atom_list[ind2].z_pos;
		
	return sqrt((x1 - x2)*(x1-x2) + (y1 - y2)*(y1 - y2) + (z1 - z2)*(z1 - z2));
}


/* 
	brute-force SDH solution in a single CPU thread 
*/
int PDH_baseline() {
	int i, j, h_pos;
	double dist;
	
	for(i = 0; i < PDH_acnt; i++) {
		for(j = i+1; j < PDH_acnt; j++) {
			dist = p2p_distance(i,j);
			h_pos = (int) (dist / PDH_res);
			histogram[h_pos].d_cnt++;
		} 
	}
	return 0;
}

/*baseline kernel function*/
__global__ void PDH_baseline_kernel(bucket *histogram, atom *atom_list, double width, int size)
{
	int i, j, h_pos;
	double distance;
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	i = x + 1;

	for(j = i; j < size; j++)
	{
		distance = p2p_distance_kernel(atom_list, x, j);
		h_pos = (int) (distance / width);
		atomicAdd(&histogram[h_pos].d_cnt, 1);
	}	
}



/* 
	set a checkpoint and show the (natural) running time in seconds 
*/
double report_running_time(int i) {
	long sec_diff, usec_diff;
	gettimeofday(&endTime, &Idunno);
	sec_diff = endTime.tv_sec - startTime.tv_sec;
	usec_diff= endTime.tv_usec-startTime.tv_usec;
	if(usec_diff < 0) {
		sec_diff --;
		usec_diff += 1000000;
	}
	if(i == 0)
		printf("Running time for CPU version: %ld.%06ld\n", sec_diff, usec_diff);
	else
		printf("Running time for GPU version: %ld.%06ld\n", sec_diff, usec_diff);

	return (double)(sec_diff*1.0 + usec_diff/1000000.0);
}


/* 
	print the counts in all buckets of the histogram 
*/
void output_histogram(bucket *histogram){
	int i; 
	long long total_cnt = 0;
	for(i=0; i< num_buckets; i++) {
		if(i%5 == 0) /* we print 5 buckets in a row */
			printf("\n%02d: ", i);
		printf("%15lld ", histogram[i].d_cnt);
		total_cnt += histogram[i].d_cnt;
	  	/* we also want to make sure the total distance count is correct */
		if(i == num_buckets - 1)	
			printf("\n T:%lld \n", total_cnt);
		else printf("| ");
	}
}

/*searches for differences between gpu and host arrays*/
void difference(bucket *a, bucket *b, bucket *c)
{
	int i, difference, found;
	difference = 0;
	//found = 0;
	for(i = 0; i <= num_buckets; i++)
	{

		if (a[i].d_cnt != b[i].d_cnt)
		{
			//found = 1;
			difference = a[i].d_cnt - b[i].d_cnt;
			if(difference < 0)
				difference = difference * -1;
			//printf("Difference detected in bucket %d: %d", i, difference);
			c[i].d_cnt+= difference;
		}
		else
		{
			c[i].d_cnt = 0;
		}

		difference = 0;	
	}
	//if(found == 0)
		//printf("NO DIFFERENCES FOUND BETWEEN HISTOGRAMS.\n");
}


//argv[1] is number of atoms and argv[2] is distance
int main(int argc, char **argv)
{
	int i;
	int hw;//indicates if CPU or GPU version is being run (0 = CPU)
	PDH_acnt = atoi(argv[1]);
	PDH_res = atof(argv[2]);
//printf("args are %d and %f\n", PDH_acnt, PDH_res);

	num_buckets = (int)(BOX_SIZE * 1.732 / PDH_res) + 1;
	histogram = (bucket *)malloc(sizeof(bucket)*num_buckets);
	atom_list = (atom *)malloc(sizeof(atom)*PDH_acnt);

	
	srand(1);
	/* generate data following a uniform distribution */
	for(i = 0;  i < PDH_acnt; i++) {
		atom_list[i].x_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
		atom_list[i].y_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
		atom_list[i].z_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
	}
	/* start counting time */
	gettimeofday(&startTime, &Idunno);
	
	/* call CPU single thread version to compute the histogram */
	PDH_baseline();
	
	/* check the total running time */ 
	hw = 0;
	report_running_time(hw);
	
	/* print out the histogram */
	output_histogram(histogram);
	
	/*ADDED CODE BELOW*/
	atom *d_x;
	bucket *d_out;
	bucket *second_hist;
	second_hist = (bucket *)malloc(sizeof(bucket)*num_buckets);
	memcpy(second_hist, histogram, sizeof(bucket)*num_buckets);
	
	/*define cuda array*/
	hipMalloc(&d_x, sizeof(atom)*PDH_acnt);//atom list
	hipMalloc(&d_out, sizeof(bucket)*num_buckets);//histogram
	//cudaMalloc(&second_hist, sizeof(bucket)*num_buckets);//histogram

	/*Copies data from host to device*/
	hipMemcpy(d_x, atom_list, sizeof(atom)*PDH_acnt, hipMemcpyHostToDevice);
	hipMemcpy(d_out, histogram, sizeof(bucket)*num_buckets, hipMemcpyHostToDevice);

	/* start counting time */
	gettimeofday(&startTime, &Idunno);

	/*launch kernel*/
	PDH_baseline_kernel<<<ceil(PDH_acnt/32), 32>>>(d_out, d_x, PDH_res, PDH_acnt);

	/* check the total running time */ 
	hw = 1;
	report_running_time(hw);

	/*copy cuda array to host array*/
	hipMemcpy(histogram, d_out, sizeof(bucket)*PDH_acnt, hipMemcpyDeviceToHost);
	
	/* print out the histogram */
	output_histogram(histogram);

	/*Print any differences*/
	printf("DIFFERENCES IN HISTOGRAMS: \n");
	bucket * diff = (bucket *)malloc(sizeof(bucket)*num_buckets);
	difference(histogram, second_hist, diff);
	output_histogram(diff);
	
	/*Free memory*/
	hipFree(d_x);
	hipFree(d_out);
	free(histogram);
	free(atom_list);
	free(second_hist);
	free(diff);
	return 0;
}







